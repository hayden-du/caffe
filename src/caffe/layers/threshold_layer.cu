#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
__global__ void ThresholdForward(const int n, const Mtype threshold,
    const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > threshold ? 1 : 0 ;
  }
}

template <typename Dtype, typename Mtype>
void ThresholdLayer<Dtype,Mtype>::Forward_gpu(const vector<BlobBase*>& bottom,
    const vector<BlobBase*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data_base<Dtype>();
  Dtype* top_data = top[0]->mutable_gpu_data_base<Dtype>();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ThresholdForward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, threshold_, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FORWARD(ThresholdLayer);
INSTANTIATE_LAYER_GPU_FORWARD_FF(ThresholdLayer);


}  // namespace caffe
