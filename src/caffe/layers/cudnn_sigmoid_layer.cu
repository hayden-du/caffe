
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
void CuDNNSigmoidLayer<Dtype,Mtype>::Forward_gpu(const vector<BlobBase*>& bottom,
    const vector<BlobBase*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data_base<Dtype>();
  Dtype* top_data = top[0]->mutable_gpu_data_base<Dtype>();
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationForward(Caffe::cudnn_handle(),
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
#else
  CUDNN_CHECK(cudnnActivationForward_v4(Caffe::cudnn_handle(),
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
#endif
}

template <typename Dtype, typename Mtype>
void CuDNNSigmoidLayer<Dtype,Mtype>::Backward_gpu(const vector<BlobBase*>& top,
    const vector<bool>& propagate_down,
    const vector<BlobBase*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  const Dtype* top_data = top[0]->gpu_data_base<Dtype>();
  const Dtype* top_diff = top[0]->gpu_diff_base<Dtype>();
  const Dtype* bottom_data = bottom[0]->gpu_data_base<Dtype>();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff_base<Dtype>();
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationBackward(Caffe::cudnn_handle(),
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
#else
  CUDNN_CHECK(cudnnActivationBackward_v4(Caffe::cudnn_handle(),
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
#endif
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNSigmoidLayer);


}  // namespace caffe
#endif
