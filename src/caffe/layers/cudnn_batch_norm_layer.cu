
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
void CuDNNBatchNormLayer<Dtype,Mtype>::Forward_gpu(
    const vector<BlobBase*>& bottom,
    const vector<BlobBase*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data_base<Dtype>();
  const void* scale_data = this->blobs_[0]->template gpu_data_base<Dtype>();
  const void* bias_data = this->blobs_[1]->template gpu_data_base<Dtype>();

  Blob<float> f_scale_data, f_bias_data;
  if (sizeof(Dtype) < 4) {
    f_scale_data.ReshapeLike(*this->blobs_[0]);
    f_bias_data.ReshapeLike(*this->blobs_[1]);
    caffe_gpu_convert(f_scale_data.count(),
        this->blobs_[0]->template gpu_data_base<Dtype>(),
        f_scale_data.mutable_gpu_data());
    caffe_gpu_convert(f_bias_data.count(),
        this->blobs_[1]->template gpu_data_base<Dtype>(),
        f_bias_data.mutable_gpu_data());
    scale_data = f_scale_data.gpu_data();
    bias_data = f_bias_data.gpu_data();
  }

  Dtype* top_data = top[0]->mutable_gpu_data_base<Dtype>();
  Dtype* save_mean = save_mean_.mutable_gpu_data();
  Dtype* save_inv_var = save_inv_var_.mutable_gpu_data();

  if (this->phase_ == TRAIN) {
    // Call Batch normalization forward
    CUDNN_CHECK(cudnnBatchNormalizationForwardTraining(
      Caffe::cudnn_handle(),
      mode_,
      cudnn::dataType<Dtype>::one,
      cudnn::dataType<Dtype>::zero,
      bottom_desc_,
      bottom_data,
      bottom_desc_,
      top_data,
      scale_bias_mean_var_desc_,
      scale_data,
      bias_data,
      1-this->moving_average_fraction_,
      this->blobs_[3]->template mutable_gpu_data_base<Dtype>(),  // mean
      this->blobs_[4]->template mutable_gpu_data_base<Dtype>(),  // variance
      epsilon_,
      save_mean,
      save_inv_var));
  } else if (this->phase_ == TEST) {
    CUDNN_CHECK(cudnnBatchNormalizationForwardInference(
      Caffe::cudnn_handle(),
      mode_,
      cudnn::dataType<Dtype>::one,
      cudnn::dataType<Dtype>::zero,
      bottom_desc_,
      bottom_data,
      bottom_desc_,
      top_data,
      scale_bias_mean_var_desc_,
      scale_data,
      bias_data,
      this->blobs_[3]->template gpu_data_base<Dtype>(),  // mean
      this->blobs_[4]->template gpu_data_base<Dtype>(),  // variance
      epsilon_));
  } else {
    LOG(FATAL) << "Unknown phase";
  }
}

template <typename Dtype, typename Mtype>
void CuDNNBatchNormLayer<Dtype,Mtype>::Backward_gpu(
    const vector<BlobBase*>& top,
    const vector<bool>& propagate_down, const vector<BlobBase*>& bottom) {
  const Dtype* top_data = top[0]->gpu_data_base<Dtype>();
  const Dtype* top_diff = top[0]->gpu_diff_base<Dtype>();
  const Dtype* bottom_data = bottom[0]->gpu_data_base<Dtype>();
  const Dtype* save_mean = save_mean_.gpu_data();
  const Dtype* save_inv_var = save_inv_var_.gpu_data();

  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff_base<Dtype>();
  const void* scale_data = this->blobs_[0]->template gpu_data_base<Dtype>();
  void* scale_diff = this->blobs_[0]->template mutable_gpu_diff_base<Dtype>();
  void* bias_diff = this->blobs_[1]->template mutable_gpu_diff_base<Dtype>();

  Blob<float> f_scale_data, f_scale_diff, f_bias_diff;
  if (sizeof(Dtype) < 4) {
    f_scale_data.ReshapeLike(*this->blobs_[0]);
    f_scale_diff.ReshapeLike(*this->blobs_[0]);
    f_bias_diff.ReshapeLike(*this->blobs_[1]);
    caffe_gpu_convert(f_scale_data.count(),
        this->blobs_[0]->template gpu_data_base<Dtype>(),
        f_scale_data.mutable_gpu_data());
    caffe_gpu_convert(f_scale_diff.count(),
        this->blobs_[0]->template gpu_diff_base<Dtype>(),
        f_scale_diff.mutable_gpu_diff());
    caffe_gpu_convert(f_bias_diff.count(),
        this->blobs_[1]->template gpu_diff_base<Dtype>(),
        f_bias_diff.mutable_gpu_diff());
    scale_data = f_scale_data.gpu_data();
    scale_diff = f_scale_diff.mutable_gpu_diff();
    bias_diff = f_bias_diff.mutable_gpu_diff();
  }

  // call Batch Normalization Backward
  CUDNN_CHECK(cudnnBatchNormalizationBackward(
      Caffe::cudnn_handle(),
      mode_,
      cudnn::dataType<Dtype>::one,
      cudnn::dataType<Dtype>::zero,
#if CUDNN_VERSION >= 4005
      cudnn::dataType<Dtype>::one,
      cudnn::dataType<Dtype>::one,
#endif
      bottom_desc_,
      bottom_data,
      bottom_desc_,
      top_diff,
      bottom_desc_,
      bottom_diff,
      scale_bias_mean_var_desc_,
      scale_data,
      scale_diff,
      bias_diff,
      this->epsilon_,
      save_mean,
      save_inv_var));

    if (sizeof(Dtype) < 4) {
      caffe_gpu_convert(f_scale_diff.count(), f_scale_diff.gpu_diff(),
          this->blobs_[0]->template mutable_gpu_diff_base<Dtype>());
      caffe_gpu_convert(f_bias_diff.count(), f_bias_diff.gpu_diff(),
          this->blobs_[1]->template mutable_gpu_diff_base<Dtype>());
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNBatchNormLayer);

}  // namespace caffe
#endif
