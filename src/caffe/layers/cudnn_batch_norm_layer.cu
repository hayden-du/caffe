
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>
#include <cudnn.h>
#include <cudnnSharedTest.h>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
void CuDNNBatchNormLayer<Dtype,Mtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const void* scale_data = this->blobs_[0]->gpu_data();
  const void* bias_data = this->blobs_[1]->gpu_data();

  cudnnTensorDescriptor_t sbdesc = scale_bias_mean_var_desc_;
  Blob<float> f_scale_data, f_bias_data;
  cudnnTensorStruct fdesc;
  if (sizeof(Dtype) < 4) {
    CUDNN_CHECK(cudnnSetTensorNdDescriptor(&fdesc,
        CUDNN_DATA_FLOAT, sbdesc->nbDims, sbdesc->dimA, sbdesc->strideA));
    sbdesc = &fdesc;
    f_scale_data.ReshapeLike(*this->blobs_[0]);
    f_bias_data.ReshapeLike(*this->blobs_[1]);
    caffe_gpu_convert(f_scale_data.count(), this->blobs_[0]->gpu_data(),
        f_scale_data.mutable_gpu_data());
    caffe_gpu_convert(f_bias_data.count(), this->blobs_[1]->gpu_data(),
        f_bias_data.mutable_gpu_data());
    scale_data = f_scale_data.gpu_data();
    bias_data = f_bias_data.gpu_data();
  }

  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* save_mean = save_mean_.mutable_gpu_data();
  Dtype* save_inv_var = save_inv_var_.mutable_gpu_data();

  if (this->phase_ == TRAIN) {
    // Call Batch normalization forward
    CUDNN_CHECK(cudnnBatchNormalizationForwardTraining(
      Caffe::cudnn_handle(),
      mode_,
      cudnn::dataType<Dtype>::one,
      cudnn::dataType<Dtype>::zero,
      bottom_desc_,
      bottom_data,
      bottom_desc_,
      top_data,
      sbdesc, //scale_bias_mean_var_desc_,
      scale_data,
      bias_data,
      1-this->moving_average_fraction_,
      this->blobs_[3]->mutable_gpu_data(),  // mean
      this->blobs_[4]->mutable_gpu_data(),  // variance
      epsilon_,
      save_mean,
      save_inv_var));
  } else if (this->phase_ == TEST) {
    CUDNN_CHECK(cudnnBatchNormalizationForwardInference(
      Caffe::cudnn_handle(),
      mode_,
      cudnn::dataType<Dtype>::one,
      cudnn::dataType<Dtype>::zero,
      bottom_desc_,
      bottom_data,
      bottom_desc_,
      top_data,
      sbdesc, //scale_bias_mean_var_desc_,
      scale_data,
      bias_data,
      this->blobs_[3]->gpu_data(),  // mean
      this->blobs_[4]->gpu_data(),  // variance
      epsilon_));
  } else {
    LOG(FATAL) << "Unknown phase";
  }
}

template <typename Dtype, typename Mtype>
void CuDNNBatchNormLayer<Dtype,Mtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* save_mean = save_mean_.gpu_data();
  const Dtype* save_inv_var = save_inv_var_.gpu_data();

  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const void* scale_data = this->blobs_[0]->gpu_data();
  void* scale_diff = this->blobs_[0]->mutable_gpu_diff();
  void* bias_diff = this->blobs_[1]->mutable_gpu_diff();

  cudnnTensorDescriptor_t sbdesc = scale_bias_mean_var_desc_;
  Blob<float> f_scale_data, f_scale_diff, f_bias_diff;
  cudnnTensorStruct fdesc;
  if (sizeof(Dtype) < 4) {
    CUDNN_CHECK(cudnnSetTensorNdDescriptor(&fdesc,
        CUDNN_DATA_FLOAT, sbdesc->nbDims, sbdesc->dimA, sbdesc->strideA));
    sbdesc = &fdesc;
    f_scale_data.ReshapeLike(*this->blobs_[0]);
    f_scale_diff.ReshapeLike(*this->blobs_[0]);
    f_bias_diff.ReshapeLike(*this->blobs_[1]);
    caffe_gpu_convert(f_scale_data.count(), this->blobs_[0]->gpu_data(),
        f_scale_data.mutable_gpu_data());
    caffe_gpu_convert(f_scale_diff.count(), this->blobs_[0]->gpu_diff(),
        f_scale_diff.mutable_gpu_diff());
    caffe_gpu_convert(f_bias_diff.count(), this->blobs_[1]->gpu_diff(),
        f_bias_diff.mutable_gpu_diff());
    scale_data = f_scale_data.gpu_data();
    scale_diff = f_scale_diff.mutable_gpu_diff();
    bias_diff = f_bias_diff.mutable_gpu_diff();
  }

  // call Batch Normalization Backward
  CUDNN_CHECK(cudnnBatchNormalizationBackward(
      Caffe::cudnn_handle(),
      mode_,
      cudnn::dataType<Dtype>::one,
      cudnn::dataType<Dtype>::zero,
      bottom_desc_,
      bottom_data,
      bottom_desc_,
      top_diff,
      bottom_desc_,
      bottom_diff,
      sbdesc, //scale_bias_mean_var_desc_,
      scale_data,
      scale_diff,
      bias_diff,
      this->epsilon_,
      save_mean,
      save_inv_var));

    if (sizeof(Dtype) < 4) {
      caffe_gpu_convert(f_scale_diff.count(), f_scale_diff.gpu_diff(),
          this->blobs_[0]->mutable_gpu_diff());
      caffe_gpu_convert(f_bias_diff.count(), f_bias_diff.gpu_diff(),
          this->blobs_[1]->mutable_gpu_diff());
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNBatchNormLayer);

}  // namespace caffe
#endif
