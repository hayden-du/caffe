#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype, typename Mtype>
__global__ void SigmoidForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = 1. / (1. + exp(-in[index])) ;
  }
}

template <typename Dtype, typename Mtype>
void SigmoidLayer<Dtype,Mtype>::Forward_gpu(const vector<BlobBase*>& bottom,
    const vector<BlobBase*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data_base<Dtype>();
  Dtype* top_data = top[0]->mutable_gpu_data_base<Dtype>();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SigmoidForward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype, typename Mtype>
__global__ void SigmoidBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    const Mtype sigmoid_x = out_data[index];
    out_diff[index] = in_diff[index] * sigmoid_x * (1 - sigmoid_x) ;
  }
}

template <typename Dtype, typename Mtype>
void SigmoidLayer<Dtype,Mtype>::Backward_gpu(const vector<BlobBase*>& top,
    const vector<bool>& propagate_down,
    const vector<BlobBase*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data_base<Dtype>();
    const Dtype* top_diff = top[0]->gpu_diff_base<Dtype>();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff_base<Dtype>();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SigmoidBackward<Dtype,Mtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidLayer);


}  // namespace caffe
